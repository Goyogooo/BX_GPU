#include "hip/hip_runtime.h"
#include <iostream> 
#include <vector>    
#include <fstream>  
#include <chrono>    
#include <algorithm> 
#include <thrust/host_vector.h>  
#include <thrust/device_vector.h> 
#include <thrust/set_operations.h> 
#include <> 
#include <hip/hip_runtime.h>             
#include <stdio.h>         
#include <hip/hip_runtime.h>   

using namespace std;

#define THREADS_PER_BLOCK 256 // ÿ��CUDA����߳���������

__device__ uint32_t* lower_bound(uint32_t* start, uint32_t* end, uint32_t value) {
    uint32_t* ptr;
    int count, step;
    count = end - start;

    while (count > 0) {
        ptr = start;
        step = count / 2;
        ptr += step;
        if (*ptr < value) {
            start = ++ptr;
            count -= step + 1;
        }
        else
            count = step;
    }
    return start;
}
__global__ void set_intersection(uint32_t* d_a, uint32_t* d_b, uint32_t* d_result, int n, int m) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        uint32_t value = d_a[i];
        uint32_t* found = lower_bound(d_b, d_b + m, value);
        if (found != (d_b + m) && *found == value) {
            d_result[i] = value;
        }
        else {
            d_result[i] = 0; // ����Ϊ����ֵ
        }
    }
}
std::vector<uint32_t> my_set_intersection(const std::vector<uint32_t>& a, const std::vector<uint32_t>& b) {
    int n = a.size();
    int m = b.size();

    uint32_t* d_a;
    uint32_t* d_b;
    uint32_t* d_result;

    hipMalloc(&d_a, n * sizeof(uint32_t));
    hipMalloc(&d_b, m * sizeof(uint32_t));
    hipMalloc(&d_result, n * sizeof(uint32_t));

    hipMemcpy(d_a, a.data(), n * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), m * sizeof(uint32_t), hipMemcpyHostToDevice);


    set_intersection <<<(n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (d_a, d_b, d_result, n, m);

    std::vector<uint32_t> result(n);
    hipMemcpy(result.data(), d_result, n * sizeof(uint32_t), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return result;
}
uint32_t read_uint32_le(std::ifstream& stream) {
    uint32_t value;
    char bytes[4];
    stream.read(bytes, 4);
    value = (static_cast<uint32_t>(static_cast<unsigned char>(bytes[3])) << 24) |
        (static_cast<uint32_t>(static_cast<unsigned char>(bytes[2])) << 16) |
        (static_cast<uint32_t>(static_cast<unsigned char>(bytes[1])) << 8) |
        static_cast<uint32_t>(static_cast<unsigned char>(bytes[0]));
    return value;
}

std::vector<uint32_t> read_array(std::ifstream& stream) {
    uint32_t length = read_uint32_le(stream);
    std::vector<uint32_t> array(length);
    for (uint32_t i = 0; i < length; ++i) {
        array[i] = read_uint32_le(stream);
    }
    return array;
}

int main() {
   /* std::ifstream file("D:/MyVS/ExpIndex", std::ios::binary);
    if (!file) {
        std::cout << "�޷��������ļ�" << std::endl;
        return 1;
    }
    
    file.seekg(32832, std::ios::beg);  
    vector<uint32_t> array1 = read_array(file);
    vector<uint32_t> array2 = read_array(file);
    file.close();*/
    std::ifstream file("D:/MyVS/ExpIndex", std::ios::binary);
    if (!file) {
        std::cerr << "�޷����ļ�" << std::endl;
        return 1;
    }
    file.seekg(32832, std::ios::beg);
    vector<uint32_t> array1 = read_array(file);
    file.seekg(1733008, std::ios::beg);
    vector<uint32_t> array2 = read_array(file);
    file.close();

    thrust::device_vector<uint32_t> d_array1 = array1;
    thrust::device_vector<uint32_t> d_array2 = array2;

   
    auto start = std::chrono::high_resolution_clock::now();

    std::vector<uint32_t> result = my_set_intersection(array1, array2);

    auto time = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start).count();
    
    std::ofstream f3("D:/MyVS/BX_GPU/result2.txt", std::ios::app);
    if (!f3.is_open()) {
        std::cerr << "�޷����ļ�" << std::endl;
        return 0;
    }
    int i = 0;
    for (uint32_t value : result) {
        if (value != 0) {
        f3 << value << ' ';
        i++;
    }
    }
    f3.close();
   
    std::cout << "����ʱ��" << time << "΢��" << " ,size:"<<i<<endl;
   
}
